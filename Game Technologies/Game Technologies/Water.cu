#include "hip/hip_runtime.h"
#include "Water.h"
#include "Renderer.h"

WaterMesh::WaterMesh(): width(100), height(100) {
	width = max(2, width);
	height = max(2, height);
	generateGrid();
	time = 0.0f;
	cudaGLSetGLDevice(1);

	if (hipGraphicsGLRegisterBuffer(&cudaVBO, bufferObject[VERTEX_BUFFER], cudaGraphicsMapFlagsNone) != hipSuccess)
		printf("Failed\n");
}

WaterMesh::~WaterMesh() {
	if (hipGraphicsUnregisterResource(cudaVBO) != hipSuccess)
		printf("Failed\n");
}

void WaterMesh::generateGrid() {
	int loop_size = 2*height + 1;

	numVertices		= width*height;
	numIndices		= (width - 1)*loop_size;

	colours			= new Vector4[numVertices];
	vertices		= new Vector3[numVertices];
	normals			= new Vector3[numVertices];
	textureCoords	= new Vector2[numVertices];
	indices			= new unsigned int[numIndices];
	
	int offset = 0;

	type = GL_TRIANGLE_STRIP;
	for (int x = 0; x < width; x++) {
		int loops = x*loop_size;
		for (int y = 0; y < height; y++) {

			if (x != width - 1)
				indices[loops + 2*y + 1] = offset;
			if (x != 0)
				indices[loops - loop_size + 2*y] = offset;

			vertices[offset]		= Vector3(2*(x*1.0f/(width-1)) - 1, 0, 2*(y*1.0f/(height-1)) - 1);
			normals[offset]			= Vector3(0, 1, 0); 
			textureCoords[offset]	= Vector2(x*1.0f/(width-1), y*1.0f/(height-1));
			colours[offset]			= Vector4(0.0f, 0.0f, 0.0f, 0.5f);
			++offset;
		}
		if (x != width - 1)
			indices[loops + loop_size - 1] = width*height;
	}

	restart_index = width*height;

	BufferData();
	glBindVertexArray(0);
}

void WaterMesh::Draw() {
	glPrimitiveRestartIndex(restart_index);
	glEnable(GL_PRIMITIVE_RESTART);
	Mesh::Draw();
	glDisable(GL_PRIMITIVE_RESTART);
}

__global__ void vboWaterResource_update(float* ptr, int width, int height, float time) {
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int offset = y*width + x;
	if (x >= width || y >= height) return;

	float period = 30;
	float rate = 0.5;

	//Center of vertex array
	float cx = x*1.0f/width - 0.5f;
	float cy = y*1.0f/height - 0.5f;

	//Wave from the center
	float wave = sin(sqrt(cx*cx + cy*cy)*period - rate*time);

	int sign = wave>0?1:-1;
	wave = sign*sqrt(sign*wave);

	//Faster waves
	period *= 3;
	rate *= -9;
	//Add two waves, in x and y direction
	ptr[3*offset + 1] = (sin(x*period/(width - 1)) + sin(y*period/(height - 1) + rate*time) - height)/2;
}

void WaterMesh::update(float msec) {
	float* devBuff;
	size_t size;
	time += msec * 0.0005;
	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks((width - 1)/threadsPerBlock.x + 1, (height - 1)/threadsPerBlock.y + 1);

	if (hipGraphicsMapResources(1, &cudaVBO, 0) != hipSuccess)
		printf("Failed\n");

	hipGraphicsResourceGetMappedPointer((void**)&devBuff, &size, cudaVBO);

	vboWaterResource_update<<<numBlocks, threadsPerBlock>>>(devBuff, width, height, time);

	if (hipGraphicsUnmapResources(1, &cudaVBO, 0) != hipSuccess)
		printf("Failed\n");
}